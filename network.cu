#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "network.h"

const int BLK_SIZE = 1024;

const char* cublasGetErrorString(hipblasStatus_t status) {
    switch(status)
    {
	case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

void fully_connected(float *input, int n_inputs, float *weights, float *biases, float *output, int n_outputs) {
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

    // Device compute
    float alpha = 1.0;
    float beta = 1.0;
    hipMemcpy(output, biases, n_outputs * sizeof(float), hipMemcpyDeviceToDevice);
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, n_outputs, n_inputs, &alpha, weights, n_outputs, input, 1, &beta, output, 1);

    hipblasDestroy(handle);
}

void convolution(float *input, int input_width, int input_height, int input_channels, float *weights, int filter_size, int num_filters, float *output, int stride, int padding) {

}

void relu(float *input, int n_inputs) {
    dim3 threads(BLK_SIZE);
    dim3 grid((int)ceil((float)n_inputs/BLK_SIZE));
    relu_kernel<<<grid, threads>>>(input, n_inputs);
}

__global__ void relu_kernel(float *input, int n_inputs) {
    int index = blockIdx.x * BLK_SIZE + threadIdx.x;

    if (index < n_inputs && input[index] < 0) {
        input[index] = 0;
    }
}

void batch_normalization(float *input, int n_inputs) {

}

void max_pooling(float *input, int input_width, int input_height, int input_channels, int filter_size, int stride, float *output) {

}
