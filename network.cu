#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"

#include "network.h"
#include "utils.h"

const int BLK_SIZE = 128;

void fully_connected(float *input, int n_inputs, float *weights, float *biases, float *output, int n_outputs) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Device compute
    float alpha = 1.0;
    float beta = 1.0;
    hipMemcpy(output, biases, n_outputs * sizeof(float), hipMemcpyDeviceToDevice);
    hipblasSgemv(handle, HIPBLAS_OP_N, n_outputs, n_inputs, &alpha, weights, n_outputs, input, 1, &beta, output, 1);

    hipblasDestroy(handle);
}

void sparse_fully_connected(SparseVector input, float *weights, float *biases, float *output, int n_outputs) {
    dim3 threads(BLK_SIZE);
    dim3 grid((int)ceil((float)n_outputs/BLK_SIZE));
    sparse_fully_connected_kernel<<<grid, threads>>>(input.values, input.indices, input.nnz, weights, biases, output, n_outputs);
}

__global__ void sparse_fully_connected_kernel(float *input_values, int *input_indices, unsigned int input_nnz, float *weights, float *biases, float *output, int n_outputs) {
    unsigned int index = blockIdx.x * BLK_SIZE + threadIdx.x;
    unsigned int element_index;

    if (index < n_outputs) {
	float ans = 0;
	for (int i = 0; i < input_nnz; i++) {
	    element_index = input_indices[i];
	    ans += input_values[i] * weights[element_index * n_outputs + index];
        }

	output[index] = ans + biases[index];
    }
}

SparseVector dsg_fully_connected(SparseVector input, float *weights, float *biases, int n_outputs, CSR projection, float sparsity) {
    // Create CUBLAS context for later
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    // Reduce dimension of weight matrix by projection x weights^T
    float alpha = 1.0 / sqrt(projection.nrows);
    float beta = 0.0;
    float *reduced_weights;
    hipMalloc(&reduced_weights, projection.nrows * n_outputs * sizeof(float));
    hipsparseHandle_t cusparse_handle;
    hipsparseCreate(&cusparse_handle);
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseScsrmm2(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, projection.nrows, n_outputs, projection.ncols, projection.nnz, &alpha, descrA, projection.values, projection.row_indx, projection.col_ids, weights, n_outputs, &beta, reduced_weights, projection.nrows);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroy(cusparse_handle);

    // Reduce dimension of input vector by projection x input
    float *reduced_input;
    hipMalloc(&reduced_input, projection.nrows * sizeof(float));
    dim3 threads(BLK_SIZE);
    dim3 grid1((int)ceil((float)projection.nrows/BLK_SIZE));
    cudaCheckErr();
    spm_spv<<<grid1, threads>>>(projection.values, projection.col_ids, projection.row_indx, input.values, input.indices, input.nnz, projection.ncols, reduced_input, projection.nrows, alpha);
    cudaCheckErr();
    
    // Approximate weights * input by (reduced weight)^T * (reduced input)
    alpha = 1.0;
    beta = 1.0;
    float *reduced_product;
    hipMalloc(&reduced_product, n_outputs * sizeof(float));
    hipMemcpy(reduced_product, biases, n_outputs * sizeof(float), hipMemcpyDeviceToDevice);
    hipblasSgemv(cublas_handle, HIPBLAS_OP_T, projection.nrows, n_outputs, &alpha, reduced_weights, projection.nrows, reduced_input, 1, &beta, reduced_product, 1);
    hipblasDestroy(cublas_handle);

    // Get min and max to estimate threshold for top values
    float *reduced_min, *reduced_max;
    hipMalloc((void**)&reduced_min, sizeof(float));
    hipMalloc((void**)&reduced_max, sizeof(float));
    dim3 grid2((int)ceil((float)n_outputs/BLK_SIZE));
    max_reduce<<<grid2, threads>>>(reduced_product, n_outputs, reduced_max);
    min_reduce<<<grid2, threads>>>(reduced_product, n_outputs, reduced_min);

    // Get list of indices of output units whose predicted activations are larger than estimated threshold
    int *significant_unit_indices;
    int *num_significant_units;
    hipMalloc(&significant_unit_indices, n_outputs * sizeof(int));
    hipMalloc((void**)&num_significant_units, sizeof(int));
    filter_activations<<<grid2, threads>>>(reduced_product, n_outputs, reduced_min, reduced_max, sparsity, significant_unit_indices, num_significant_units);
  
    // Multiply weights with input vector according to binary mask
    SparseVector output;
    hipMemcpy(&output.nnz, num_significant_units, sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&output.values, output.nnz * sizeof(float));
    hipMalloc(&output.indices, output.nnz * sizeof(int));
    hipMemcpy(output.indices, significant_unit_indices, output.nnz * sizeof(int), hipMemcpyDeviceToDevice);
    dim3 grid3((int)ceil((float)output.nnz/BLK_SIZE));
    filtered_product<<<grid2, threads>>>(weights, input.values, input.indices, input.nnz, biases, n_outputs, significant_unit_indices, num_significant_units, output.values);

    // Add freeing stuff up here 
    hipFree(reduced_weights);
    hipFree(reduced_input);
    hipFree(reduced_product);
    hipFree(reduced_min);
    hipFree(reduced_max);
    hipFree(significant_unit_indices);
    hipFree(num_significant_units);

    return output;
}

__global__ void spm_spv(float *mat_values, int *mat_col_ids, int *mat_row_indx, float *vec_values, int *vec_indices, int vec_nnz, int vec_len, float *output, int output_len, float alpha) {
    unsigned int index = blockIdx.x * BLK_SIZE + threadIdx.x;
    float ans = 0;
    __shared__ float s_vec_values[BLK_SIZE];
    __shared__ int s_vec_indices[BLK_SIZE];

    int mat_row_end = index < output_len ? mat_row_indx[index + 1] : 0;

    int mat_pos = index <= output_len ? mat_row_indx[index] : 0;
    int vec_pos = 0;
    int mat_col, vec_row;
    int vec_val_index;
    int mat_col_end;

    // We load the vector values and indices into shared memory in intervals
    int num_intervals = ceil((float)vec_nnz / BLK_SIZE);
    for (int interval = 0; interval < num_intervals; interval++) {
	vec_val_index = interval * BLK_SIZE + threadIdx.x;
	s_vec_values[threadIdx.x] = vec_val_index < vec_nnz ? vec_values[vec_val_index] : 0;
	s_vec_indices[threadIdx.x] = vec_val_index < vec_nnz ? vec_indices[vec_val_index] : vec_len;
	__syncthreads();

	// This loop calculates the dot product of the vector interval and the corresponding matrix row interval
	vec_pos = 0;
        mat_col_end = s_vec_indices[BLK_SIZE - 1];
        mat_col = mat_pos < mat_row_end ? mat_col_ids[mat_pos] : vec_len;

	while (mat_col <= mat_col_end && vec_pos < BLK_SIZE) {
	    vec_row = s_vec_indices[vec_pos];

	    if (mat_col == vec_row) {
	        ans += mat_values[mat_pos] * s_vec_values[vec_pos];
		mat_pos += 1;
		vec_pos += 1;
	    } else if (mat_col < vec_row) {
		mat_pos += 1;
	    } else {
		vec_pos += 1;
	    }
	    mat_col = mat_pos < mat_row_end ? mat_col_ids[mat_pos] : vec_len + 1;
        }
	    
        __syncthreads();
    }

    if (index < vec_len)
        output[index] = ans * alpha;
    
}

__device__ float atomicMaxf(float *address, float val) {
    int *address_as_int = (int*) address;
    int old = *address_as_int, assumed;
    do {
	assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}
__device__ float atomicMinf(float *address, float val) {
    int *address_as_int = (int*) address;
    int old = *address_as_int, assumed;
    do {
	assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void max_reduce(float *values, int num_elements, float *max) {
    __shared__ float shared_max[BLK_SIZE];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid == 0)
	*max = -FLT_MAX;

    shared_max[tid] = -FLT_MAX;
    if (gid < num_elements) 
	shared_max[tid] = values[gid];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
	if (tid < s && gid < num_elements) 
	    shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid + s]);
	__syncthreads();
    }

    if (tid == 0) 
	atomicMaxf(max, shared_max[0]);
}

__global__ void min_reduce(float *values, int num_elements, float *min) {
    __shared__ float shared_min[BLK_SIZE];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid == 0)
	*min = FLT_MAX;

    shared_min[tid] = FLT_MAX;
    if (gid < num_elements) 
	shared_min[tid] = values[gid];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
	if (tid < s && gid < num_elements) 
	    shared_min[tid] = fminf(shared_min[tid], shared_min[tid + s]);
	__syncthreads();
    }

    if (tid == 0) 
	atomicMinf(min, shared_min[0]);
}

__global__ void filter_activations(float *values, int num_elements, float *min, float *max, float sparsity, int *significant_unit_indices, int *num_significant_units) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int l_numfiltered_tb, g_pos;
    __shared__ float threshold;
    int l_pos = -1;

    float val = -FLT_MAX;
    if (id < num_elements) 
        val = values[id];

    if (id == 0)
	*num_significant_units = 0;

    // Init local counter and threshold
    if (threadIdx.x == 0) {
	l_numfiltered_tb = 0;
        threshold = (*max + *min) / 2.0 + (*max - *min) / M_PI * asinf(2 * sparsity - 1);
    }
    __syncthreads();

    // Evaluate threshold comparison
    if (val > threshold)
	l_pos = atomicAdd(&l_numfiltered_tb, 1);
    __syncthreads();

    // Get global index
    if(threadIdx.x == 0)
	g_pos = atomicAdd(num_significant_units, l_numfiltered_tb);
    __syncthreads();

    if (l_pos != -1)
	significant_unit_indices[g_pos + l_pos] = id;
}

__global__ void filtered_product(float *weights, float *input_values, int *input_indices, int input_nnz, float *biases, int n_outputs, int *significant_unit_indices, int *num_significant_units, float *output_values) {
    unsigned int val_index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float s_input_values[BLK_SIZE];
    __shared__ float s_input_indices[BLK_SIZE];
    
    int unit_index = val_index < *num_significant_units ? significant_unit_indices[val_index] : 0;
    int input_index = 0;
    double ans = 0;
    int input_val_index = 0;
    
    // We load the input_values into shared memory in intervals
    int num_intervals = (int)ceil((float)input_nnz/BLK_SIZE);
    for (int interval = 0; interval < num_intervals; interval++) {
	input_val_index = interval * BLK_SIZE + threadIdx.x;
	s_input_values[threadIdx.x] = input_val_index < input_nnz ? input_values[input_val_index] : 0;
        s_input_indices[threadIdx.x] = input_val_index < input_nnz ? input_indices[input_val_index] : 0;
	__syncthreads();

        for (int i = 0; i < BLK_SIZE; i++) {
            input_index = s_input_indices[i];
	    ans += weights[input_index * n_outputs + unit_index] * s_input_values[i];
        }
	__syncthreads();
    }

    if (val_index < *num_significant_units) {
        output_values[val_index] = ans + biases[unit_index];
    }
}

void relu(float *input, int n_inputs) {
    dim3 threads(BLK_SIZE);
    dim3 grid((int)ceil((float)n_inputs/BLK_SIZE));
    relu_kernel<<<grid, threads>>>(input, n_inputs);
}

__global__ void relu_kernel(float *input, int n_inputs) {
    int index = blockIdx.x * BLK_SIZE + threadIdx.x;

    if (index < n_inputs && input[index] < 0) {
        input[index] = 0;
    }
}

void batch_normalization(float *input, int n_inputs) {

}

