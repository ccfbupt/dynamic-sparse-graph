#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"

#include "network.h"

const int BLK_SIZE = 1024;

const char* cublasGetErrorString(hipblasStatus_t status) {
    switch(status)
    {
	case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

const char* hipsparseGetErrorString(hipsparseStatus_t status) {
    switch(status)
    {
	case HIPSPARSE_STATUS_SUCCESS: return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED: return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED: return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE: return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH: return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR: return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED: return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR: return "HIPSPARSE_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

void fully_connected(float *input, int n_inputs, float *weights, float *biases, float *output, int n_outputs) {
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

    // Device compute
    float alpha = 1.0;
    float beta = 1.0;
    hipMemcpy(output, biases, n_outputs * sizeof(float), hipMemcpyDeviceToDevice);
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, n_outputs, n_inputs, &alpha, weights, n_outputs, input, 1, &beta, output, 1);

    hipblasDestroy(handle);
}

void sparse_fully_connected(SparseVector input, float *weights, float *biases, float *output, int n_outputs) {
    dim3 threads(BLK_SIZE);
    dim3 grid((int)ceil((float)n_outputs/BLK_SIZE));
    sparse_fully_connected_kernel<<<grid, threads>>>(input.values, input.indices, input.nnz, weights, biases, output, n_outputs);
}

__global__ void sparse_fully_connected_kernel(float *input_values, int *input_indices, unsigned int input_nnz, float *weights, float *biases, float *output, int n_outputs) {
    unsigned int index = blockIdx.x * BLK_SIZE + threadIdx.x;
    unsigned int element_index;

    if (index < n_outputs) {
	float ans = 0;
	for (int i = 0; i < input_nnz; i++) {
	    element_index = input_indices[i];
	    ans += input_values[i] * weights[element_index * n_outputs + index];
        }

	output[index] = ans + biases[index];
    }
}

void dsg_fully_connected(SparseVector input, float *weights, float *biases, float *output, int n_outputs, CSR projection, float sparsity) {
    // Create CUBLAS context for later
    hipblasHandle_t dense_handle;
    hipblasCreate(&dense_handle);

    // Reduce dimension of weight matrix by projecting it with projection
    float alpha = 1.0 / sqrt(projection.nrows);
    float beta = 0.0;
    float *reduced_weights;
    hipMalloc(&reduced_weights, projection.nrows * n_outputs * sizeof(float));
    hipsparseHandle_t sparse_handle;
    hipsparseCreate(&sparse_handle);
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseScsrmm2(sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, projection.nrows, n_outputs, projection.ncols, projection.nnz, &alpha, descrA, projection.values, projection.row_indx, projection.col_ids, weights, n_outputs, &beta, reduced_weights, projection.nrows);
    hipsparseDestroy(sparse_handle);
    hipsparseDestroyMatDescr(descrA);

    // Reduce dimension of input vector by projecting it with projection
    float *reduced_input;
    hipMalloc(&reduced_input, projection.nrows * sizeof(float));
    dim3 threads(BLK_SIZE);
    dim3 grid((int)ceil((float)projection.nrows/BLK_SIZE));
    spm_spv<<<grid, threads>>>(projection.values, projection.col_ids, projection.row_indx, input.values, input.indices, input.nnz, reduced_input, projection.nrows, alpha);

    // Multiply reduced weight with reduced vector
    alpha = 1.0;
    beta = 1.0;
    float *reduced_product;
    hipMalloc(&reduced_product, n_outputs * sizeof(float));
    hipMemcpy(reduced_product, biases, n_outputs * sizeof(float), hipMemcpyDeviceToDevice);
    hipblasSgemv(dense_handle, HIPBLAS_OP_T, n_outputs, projection.nrows, &alpha, reduced_weights, n_outputs, reduced_input, 1, &beta, reduced_product, 1);
    hipblasDestroy(dense_handle);
    
    // Top k search to find output units with large approximate activation
    int K = (int)ceil(n_outputs * (1.0 - sparsity));
    int *top_indices = (int*) malloc(K * sizeof(int));
    
}

__global__ void spm_spv(float *mat_values, int *mat_col_ids, int *mat_row_indx, float *vec_values, int *vec_indices, int vec_nnz, float *output, int output_len, float alpha) {
    unsigned int index = blockIdx.x * BLK_SIZE + threadIdx.x;
    float ans = 0;

    if (index < output_len) {
        int mat_row_end = mat_row_indx[index + 1];

        int mat_pos = mat_row_indx[index];
        int vec_pos = 0;
        int mat_col, vec_row;

	while (mat_pos < mat_row_end && vec_pos < vec_nnz) {
	    mat_col = mat_col_ids[mat_pos];
	    vec_row = vec_indices[vec_pos];
	    if (index == 1)
		printf("%d %d %f %d %d %f\n", mat_pos, mat_col, mat_values[mat_pos], vec_pos, vec_row, vec_values[vec_pos]);
	    if (mat_col == vec_row) {
		ans += mat_values[mat_pos] * vec_values[vec_pos];
		mat_pos += 1;
		vec_pos += 1;
	    } else if (mat_col < vec_row) {
		mat_pos += 1;
	    } else {
		vec_pos += 1;
	    }
        }

        output[index] = ans * alpha;
    }
}

void relu(float *input, int n_inputs) {
    dim3 threads(BLK_SIZE);
    dim3 grid((int)ceil((float)n_inputs/BLK_SIZE));
    relu_kernel<<<grid, threads>>>(input, n_inputs);
}

__global__ void relu_kernel(float *input, int n_inputs) {
    int index = blockIdx.x * BLK_SIZE + threadIdx.x;

    if (index < n_inputs && input[index] < 0) {
        input[index] = 0;
    }
}

void batch_normalization(float *input, int n_inputs) {

}

