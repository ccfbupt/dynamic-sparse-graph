#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "network.h"

const int BLK_SIZE = 1024;

void fully_connected(float *input, int n_inputs, float *weights, float *output, int n_outputs) {
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Device alloc
    float *d_weights, *d_input, *d_output;
    hipMalloc((void**)&d_weights, n_outputs * n_inputs * sizeof(float));
    hipMalloc((void**)&d_input, n_inputs * sizeof(float));
    hipMalloc((void**)&d_output, n_outputs * sizeof(float));

    // Copy to device
    stat = hipblasSetMatrix(n_outputs, n_inputs, sizeof(float), weights, n_outputs, d_weights, n_outputs);
    stat = hipblasSetVector(n_inputs, sizeof(float), input, 1, d_input, 1);
    stat = hipblasSetVector(n_outputs, sizeof(float), output, 1, d_output, 1);

    // Device compute
    float alpha = 1.0;
    float beta = 0.0;
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, n_outputs, n_inputs, &alpha, d_weights, n_outputs, d_input, 1, &beta, d_output, 1);

    // Copy answer from device
    stat = hipblasGetVector(n_outputs, sizeof(float), d_output, 1, output, 1);

    // Clean up
    hipFree(d_weights);
    hipFree(d_input);
    hipFree(d_output);
    hipblasDestroy(handle);
}

void convolution(float *input, int input_width, int input_height, int input_channels, float *weights, int filter_size, int num_filters, float *output, int stride, int padding) {

}

__global__ void relu(float *input, int n_inputs) {
    int index = blockIdx.x * BLK_SIZE + threadIdx.x;

    if (index < n_inputs && input[index] < 0) {
        input[index] = 0;
    }
}

void batch_normalization(float *input, int n_inputs) {

}

void max_pooling(float *input, int input_width, int input_height, int input_channels, int filter_size, int stride, float *output) {

}
