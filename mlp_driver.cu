#include <iostream>
#include <fstream>
#include <math.h>
#include <time.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"

#include "network.h"
#include "utils.h"

const int num_images = 60000;
const int image_len = 28 * 28;
const int msg_freq = 5000;

const int n_inputs = 784;
const int n_hidden1 = 256;
const int n_hidden2 = 256;
const int n_outputs = 10;

float sparsity = 0.5;

float dense_infer(MLP mlp, float *input_data, unsigned char *labels) {

    printf("Starting dense inference...\n");

    // Declare network activations
    float *d_input, *d_fc1_activations, *d_fc2_activations, *d_net_output;
    hipMalloc(&d_input, n_inputs * sizeof(float));
    hipMalloc(&d_fc1_activations, n_hidden1 * sizeof(float));
    hipMalloc(&d_fc2_activations, n_hidden2 * sizeof(float));
    hipMalloc(&d_net_output, n_outputs * sizeof(float));

    float *net_output = (float*) malloc(n_outputs * sizeof(float));
    int num_correct = 0;
    unsigned char pred = 0;
    float max_output = -1000;

    int test_size = 60000;
    for (int n = 0; n < test_size; n++) {

	// Copy input to gpu
	hipMemcpy(d_input, input_data + n * image_len, image_len * sizeof(float), hipMemcpyHostToDevice);

        // Device compute
        fully_connected(d_input, n_inputs, mlp.fc1_weights, mlp.fc1_biases, d_fc1_activations, n_hidden1);
        relu(d_fc1_activations, n_hidden1);
        fully_connected(d_fc1_activations, n_hidden1, mlp.fc2_weights, mlp.fc2_biases, d_fc2_activations, n_hidden2);
        relu(d_fc2_activations, n_hidden2);
        fully_connected(d_fc2_activations, n_hidden2, mlp.fc3_weights, mlp.fc3_biases, d_net_output, n_outputs);

        // Copy network output to host
        hipMemcpy(net_output, d_net_output, n_outputs * sizeof(float), hipMemcpyDeviceToHost);

        pred = 0;
        max_output = 0;
        for (unsigned char i = 0; i < n_outputs; i++) {
            if (net_output[i] > max_output) {
                pred = i;
                max_output = net_output[i];
            }
        }

        if (pred == labels[n]) 
	    num_correct++;

        if ((n + 1) % msg_freq == 0)
            printf("Done with %d instances.\n", n + 1);
    }

    // Clean up
    free(net_output);
    hipFree(d_input);
    hipFree(d_fc1_activations);
    hipFree(d_fc2_activations);
    hipFree(d_net_output);

    printf("Finished dense inference.\n");

    return (float)num_correct / test_size;
}


float dsg_infer(MLP mlp, float *input_data, unsigned char *labels, float sparsity) {
    printf("Starting dsg inference...\n");

    // Declare network activations and output
    SparseVector s_input, s_fc1_activations, s_fc2_activations;
    float *net_output = (float*) malloc(n_outputs * sizeof(float));
    float *d_net_output;
    hipMalloc(&d_net_output, n_outputs * sizeof(float));

    int num_correct = 0;
    unsigned char pred = 0;
    float max_output = -FLT_MAX;

    // Generate linear projections for each layer
    CSR projection1 = get_random_projection(150, n_inputs, 3, 1);
    CSR projection2 = get_random_projection(50, n_hidden1, 3, 1);

    int test_size = 10000;

    struct SparseVector inputs[test_size];
    for (int n = 0; n < test_size; n++) {
	inputs[n] = dense_to_SparseVector(input_data + n * image_len, image_len, 1);
    }

    for (int n = 0; n < test_size; n++) {

        // Copy network input to gpu
        //s_input = dense_to_SparseVector(input_data + n * image_len, image_len, 1);

        // Device compute
        s_fc1_activations = dsg_fully_connected(inputs[n], mlp.fc1_weights, mlp.fc1_biases, n_hidden1, projection1, sparsity);
        relu(s_fc1_activations.values, s_fc1_activations.nnz);
        s_fc2_activations = dsg_fully_connected(s_fc1_activations, mlp.fc2_weights, mlp.fc2_biases, n_hidden2, projection2, sparsity);
        relu(s_fc2_activations.values, s_fc2_activations.nnz);
        sparse_fully_connected(s_fc2_activations, mlp.fc3_weights, mlp.fc3_biases, d_net_output, n_outputs);

        // Copy network output to host
        hipMemcpy(net_output, d_net_output, n_outputs * sizeof(float), hipMemcpyDeviceToHost);

	// Compare network output with true label
        pred = 0;
        max_output = 0;
        for (unsigned char i = 0; i < n_outputs; i++) {
            if (net_output[i] > max_output) {
                pred = i;
                max_output = net_output[i];
            }
        }

        if (pred == labels[n])
            num_correct++;

        if ((n + 1) % msg_freq == 0)
            printf("Done with %d instances.\n", n + 1);
    }

    // Clean up
    free(net_output);
    hipFree(s_input.values);
    hipFree(s_input.indices);
    hipFree(s_fc1_activations.values);
    hipFree(s_fc1_activations.indices);
    hipFree(projection1.values);
    hipFree(projection1.col_ids);
    hipFree(projection1.row_indx);
    hipFree(projection2.values);
    hipFree(projection2.col_ids);
    hipFree(projection2.row_indx);

    printf("Finished dsg inference.\n");

    return (float)num_correct / test_size;
}

int main(int argc, char *argv[]) {

    // Load model parameters
    MLP mlp = load_mlp("models/mlp_weights.bin", n_inputs, n_hidden1, n_hidden2, n_outputs);

    // Declare and load in data
    float *input_data = (float*) malloc(num_images * image_len * sizeof(float));
    load_mnist_data(input_data);

    // Declare and load in labels
    unsigned char *labels = (unsigned char*) malloc(num_images * sizeof(unsigned char));
    load_mnist_labels(labels);

    //float dense_accuracy = dense_infer(mlp, input_data, labels);
    //printf("Dense accuracy: %f\n\n", dense_accuracy);
    float dsg_accuracy = dsg_infer(mlp, input_data, labels, sparsity);
    printf("DSG accuracy: %f\n\n", dsg_accuracy);

    // Clean up
    free(input_data);
    free(labels);

    hipFree(mlp.fc1_weights);
    hipFree(mlp.fc1_biases);
    hipFree(mlp.fc2_weights);
    hipFree(mlp.fc2_biases);
    hipFree(mlp.fc3_weights);
    hipFree(mlp.fc3_biases);

    return 0;
}

